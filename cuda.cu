#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "cudaHeader.h"
#include <hip/hip_runtime_api.h>

#define MAX_FIRST_TYPE_GROUPS_ROW_SIZE 9
#define MAX_FIRST_TYPE_GROUPS_COL_SIZE 5
#define MAX_SECOND_TYPE_GROUPS_ROW_SIZE 11
#define MAX_SECOND_TYPE_GROUPS_COL_SIZE 7

//COMPARE 2 LETTERS TO CHECK IF THEY ARE FROM FIRST TYPE GROUP
__device__ int search_first_type_groups(char char1, char char2)
{
    char first_type_groups[MAX_FIRST_TYPE_GROUPS_ROW_SIZE][MAX_FIRST_TYPE_GROUPS_COL_SIZE] = { "NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK", "FYW", "HY", "MILF" };
	int counter = 0;
	for (int i = 0; i < MAX_FIRST_TYPE_GROUPS_ROW_SIZE; i++)
	{
		for (int j = 0;j < MAX_FIRST_TYPE_GROUPS_COL_SIZE; j++)
		{
			if (first_type_groups[i][j] == char1)
				counter++;
			if (first_type_groups[i][j] == char2)
				counter++;
			if (counter == 2)
				return 1;
		}
		counter = 0;
	}
	return 0;
}

//COMPARE 2 LETTERS TO CHECK IF THEY ARE FROM SECOND TYPE GROUP
__device__ int search_second_type_groups(char char1, char char2)
{
    char second_type_groups[MAX_SECOND_TYPE_GROUPS_ROW_SIZE][MAX_SECOND_TYPE_GROUPS_COL_SIZE] = { "SAG", "ATV", "CSA", "SGND", "STPA", "STNK", "NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM" };
	int counter = 0;
	for (int i = 0; i < MAX_SECOND_TYPE_GROUPS_ROW_SIZE; i++)
	{
		for (int j = 0;j < MAX_SECOND_TYPE_GROUPS_COL_SIZE; j++)
		{
			if (second_type_groups[i][j] == char1)
				counter++;
			if (second_type_groups[i][j] == char2)
				counter++;
			if (counter == 2)
				return 1;
		}
		counter = 0;
	}
	return 0;
}

//RETURN THE APPROPRIATE WEIGHT
__device__ int calculate_score_between_two_characters(char c1, char c2, int* weights)
{
	if (c1 == c2)
		return weights[0];
	else if(search_first_type_groups(c1, c2) == 1)
		return (0-weights[1]);
	else if(search_second_type_groups(c1, c2) == 1)
		return (0-weights[2]);
	else 
		return (0-weights[3]);																																							
}

__global__ void calculate_score(char* cuda_seq1,char* cuda_sequance,int* cuda_res,int offset_mutant,int start_offset, int* weights)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	char seq1 = cuda_seq1[start_offset + index];
	char seq2 = cuda_sequance[index];
	
	if(index == offset_mutant)
	{	
		cuda_res[index] = 0;
	}
	else
	{
		if(index > offset_mutant)
		{
			seq2 = cuda_sequance[index - 1];
		}
		cuda_res[index] = calculate_score_between_two_characters(seq1, seq2, weights);
	}
}

int compute_on_gpu(char* seq1,char* sequences,int weights[],int offset,int offset_mutant,int length_offset_mutant, int* results)
{
	hipError_t error = hipSuccess;
	//ALLOCATIONS AND COPYING DATA TO CUDA
	size_t size_sequance = (strlen(sequences) + 1)*(sizeof(char));
	char* cuda_sequance;
	error = hipMalloc((void**)&cuda_sequance, size_sequance);
	if(error != hipSuccess)
	{
		fprintf(stderr,"failed to allocate memory %s\n",hipGetErrorString(error));
	}
	error = hipMemcpy(cuda_sequance,sequences,size_sequance,hipMemcpyHostToDevice);
	if(error != hipSuccess)
	{
		fprintf(stderr,"failed to copy memory %s\n",hipGetErrorString(error));
	}
	
	size_t size_seq1 = (strlen(seq1) + 1)*(sizeof(char));
	char* cuda_seq1;
	error = hipMalloc((void**)&cuda_seq1, size_seq1);
	if(error != hipSuccess)
	{
		fprintf(stderr,"failed to allocate memory %s\n",hipGetErrorString(error));
	}
	error = hipMemcpy(cuda_seq1,seq1,size_seq1,hipMemcpyHostToDevice);
	if(error != hipSuccess)
	{
		fprintf(stderr,"failed to copy memory %s\n",hipGetErrorString(error));
	}
	
	size_t size_res = (strlen(sequences) + 1)*(sizeof(int));
	int* cuda_res;
	error = hipMalloc((void**)&cuda_res, size_res);
	if(error != hipSuccess)
	{
		fprintf(stderr,"failed to allocate memory %s\n",hipGetErrorString(error));
	}
	error = hipMemcpy(cuda_res,results,size_res,hipMemcpyHostToDevice);
	if(error != hipSuccess)
	{
		fprintf(stderr,"failed to copy memory %s\n",hipGetErrorString(error));
	}
	
	size_t size_weights = 4 * sizeof(int);
	int* cuda_w;
	error = hipMalloc((void**)&cuda_w, size_weights);
	if(error != hipSuccess)
	{
		fprintf(stderr,"failed to allocate memory %s\n",hipGetErrorString(error));
	}
	error = hipMemcpy(cuda_w,weights,size_weights,hipMemcpyHostToDevice);
	if(error != hipSuccess)
	{
		fprintf(stderr,"failed to copy memory %s\n",hipGetErrorString(error));
	}
	//UPDATE THE SCORE FOR THE WORST CASE
	int max_score = strlen(sequences) *  -(weights[1] + weights[2] + weights[3]);
	int index_mutant = 0;
	
	for(int i = offset_mutant;i < (offset_mutant + length_offset_mutant);i++)
	{
		int number_element = (strlen(sequences) + 1) / 1024;
		int block_per_grid = number_element + 1;
		int thread_per_block = (strlen(sequences)+ 1) / block_per_grid;
		calculate_score<<<block_per_grid, thread_per_block>>>(cuda_seq1,cuda_sequance,cuda_res, i, offset, cuda_w);
		
		error = hipMemcpy(results,cuda_res,size_res,hipMemcpyDeviceToHost);
		if(error != hipSuccess)
		{
			fprintf(stderr,"failed to copy memory %s\n",hipGetErrorString(error));
		}

		int score = 0;
		for(int j = 0; j < strlen(sequences) + 1; j++)
		{
			score += results[j];
		}
		//UPDATE THE SCORE TO MAX SCORE
		if(score > max_score)
		{
			max_score = score;
			index_mutant = i;
		}
	}
	//SAVE THE FINAL SCORE IN A KNOWN PLACE
	results[0] = max_score;
	//FREE ALLOCATIONS IN CUDA
	if (hipFree(cuda_sequance) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	if (hipFree(cuda_seq1) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	if (hipFree(cuda_res) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	if (hipFree(cuda_w) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	return index_mutant;
}
